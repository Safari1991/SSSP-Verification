#include "hip/hip_runtime.h"
/***********************************************************************************
Created by Mohsen Safari.
************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>


//@ pure int inf() = -1; //100000000; any negative integer

/*@ 
pure bool Graph(int V, int A, seq<int> start, seq<int> end, seq<int> weight) =
	0 < V &&
	0 < A && |start| == A && |end| == A && |weight| == A &&
	(\forall int i; 0 <= i && i < A; 
		0 <= start[i] && start[i] < V && 
		0 <= end[i] && end[i] < V && 
		start[i] != end[i] && 
		(\forall int j; 0 <= j && j < A && i != j && start[i] == start[j]; end[i] != end[j]) &&
     weight[i] > 0); 
@*/

/*@ 
requires Graph(V, A, start, end, weight);
ensures (\forall int i; i >= 0 && i < A; Path(V, A, start, end, weight, start[i], end[i], seq<int> { i }));
pure bool Path(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) =
	0 <= x && x < V &&
	0 <= y && y < V &&
	(\forall int i; 0 <= i && i < |P|; 0 <= P[i] && P[i] < A) &&
	(0 == |P| ==> x == y) &&
	(0 < |P| ==> start[P[0]] == x && end[P[|P| - 1]] == y) &&
	(\forall int i; 0 <= i && i < |P| - 1; end[P[i]] == start[P[i + 1]]);
@*/

/*@ 
requires Graph(V, A, start, end, weight);
ensures (\forall int i; i >= 0 && i < A; SimplePath(V, A, start, end, weight, start[i], end[i], seq<int> { i }));
pure bool SimplePath(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) =
  Path(V, A, start, end, weight, x, y, P) &&
  (\forall int i; 0 <= i && i < |P| - 1; (\forall int j; i < j && j < |P|; start[P[i]] != end[P[j]])); 
@*/

/*@ 
requires Graph(V, A, start, end, weight);
pure bool ExPath(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, int len) =
	(\exists seq<int> P; Path(V, A, start, end, weight, x, y, P); |P| <= len);
@*/

/*@   
requires Graph(V, A, start, end, weight);
ensures (\forall int i; i >= 0 && i < A; ExPathEqual(V, A, start, end, weight, start[i], end[i], weight[i]));
ensures ExPathEqual(V, A, start, end, weight, x, x, 0);
pure bool ExPathEqual(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, int len) =
	(\exists seq<int> P; Path(V, A, start, end, weight, x, y, P); cost(V, A, start, end, weight, P) == len);
@*/

/*@
requires Graph(V, A, start, end, weight);
requires (\forall int i; 0 <= i && i < |P|; 0 <= P[i] && P[i] < A);
ensures 0 <= \result;
ensures 0 == |P| ==> \result == 0;
ensures (P != seq<int>{}) ==> \result > 0;
ensures (\result == 0) ==> (P == seq<int>{});
ensures (\result != 0) ==> (P != seq<int>{});
ensures 1 == |P| ==> \result == weight[P[0]];
pure int cost(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> P) =  
  0 < |P| ? weight[P[0]] + cost(V, A, start, end, weight, tail(P)) : 0;
@*/  

/*@
ensures (xs + ys) + zs == xs + (ys + zs);
pure bool iseq_assoc(seq<int> xs, seq<int> ys, seq<int> zs) = true;
@*/

/*@
requires Graph(V, A, start, end, weight);
requires Path(V, A, start, end, weight, x, y, P);
requires 0 <= a && a < A;
requires end[a] == x;
ensures \result && Path(V, A, start, end, weight, start[a], y, seq<int> { a } + P);
pure bool lemma_path_append(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P, int a);
@*/

/*@
requires Graph(V, A, start, end, weight);
requires Path(V, A, start, end, weight, s, t, P);
requires Path(V, A, start, end, weight, t, u, Q);
ensures \result && Path(V, A, start, end, weight, s, u, P + Q);
pure bool lemma_path_trans(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int s, int t, int u, seq<int> P, seq<int> Q) =
	0 < |P| ==> (
		lemma_path_trans(V, A, start, end, weight, end[P[0]], t, u, tail(P), Q) &&
		lemma_path_append(V, A, start, end, weight, end[P[0]], u, tail(P) + Q, P[0]) &&
		iseq_assoc(seq<int> { P[0] }, tail(P), Q) &&
		seq<int> { P[0] } + tail(P) == P
	); 
@*/

/*@
requires 0 < |xs|;
ensures (xs + ys)[0] == xs[0];
ensures tail(xs + ys) == tail(xs) + ys;
ensures \result;  
pure bool lemma_app_nonempty(seq<int> xs, seq<int> ys) = true; 
@*/

/*@
requires 0 < |ys|;
ensures \result && ((xs + seq<int>{ys[0]}) + tail(ys)) == xs + ys;
pure bool lemma_app_append_right(seq<int> xs, seq<int> ys) =
  iseq_assoc(xs, seq<int>{ys[0]}, tail(ys));
@*/

/*@
requires Graph(V, A, start, end, weight);
requires (\forall int i; 0 <= i && i < |P|; 0 <= P[i] && P[i] < A);
requires (\forall int i; 0 <= i && i < |Q|; 0 <= Q[i] && Q[i] < A);
ensures \result && cost(V, A, start, end, weight, P + Q) == cost(V, A, start, end, weight, P) + cost(V, A, start, end, weight, Q);
pure bool lemma_cost_app(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> P, seq<int> Q) =  
  0 < |P| ==> lemma_app_nonempty(P, Q) && lemma_cost_app(V, A, start, end, weight, tail(P), Q);
@*/

/*@
requires Graph(V, A, start, end, weight);
requires Path(V, A, start, end, weight, s, t, P);
requires Path(V, A, start, end, weight, t, u, Q);
ensures \result && Path(V, A, start, end, weight, s, u, P + Q);
ensures \result && cost(V, A, start, end, weight, P + Q) == cost(V, A, start, end, weight, P) + cost(V, A, start, end, weight, Q);
pure bool lemma_cost_path(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int s, int t, int u, seq<int> P, seq<int> Q) =
  lemma_path_trans(V, A, start, end, weight, s, t, u, P, Q) &&
  lemma_cost_app(V, A, start, end, weight, P, Q);
@*/

/*@
requires b;
ensures \result == xs;
pure seq<int> assertSeq(seq<int> xs, bool b) = xs;
@*/

/*@
requires Graph(V, A, start, end, weight);
ensures \result && (\forall int i; i >= 0 && i < A; Path(V, A, start, end, weight, start[i], end[i], seq<int>{i}));
ensures	\result && (\forall int i; i >= 0 && i < A; ExPathEqual(V, A, start, end, weight, start[i], end[i], weight[i]));	
pure bool lemma_one_path(int V, int A, seq<int> start, seq<int> end, seq<int> weight) = true;
@*/


/* ** Invariant 10 */
/*@
requires Graph(V, A, start, end, weight);
requires ExPathEqual(V, A, start, end, weight, s, t, len1); 
requires ExPathEqual(V, A, start, end, weight, t, u, len2); 
ensures \result && ExPathEqual(V, A, start, end, weight, s, u, len1+len2);
pure bool lemma_expath_trans(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int s, int t, int u, int len1, int len2) =
	(\forall seq<int> P; Path(V, A, start, end, weight, s, t, P);
		(\forall seq<int> Q; Path(V, A, start, end, weight, t, u, Q);
			lemma_cost_path(V, A, start, end, weight, s, t, u, P, Q)));
@*/


/*@
  requires i >= 0 && i <= |xs|-1;
  ensures |\result| == |xs| - 1 - i;
  ensures (\forall int k; k >= 0 && k < |\result|; \result[k] == xs[i+k]);
  ensures i == 0 ==> (\forall int k; k >= 0 && k < |xs|-1; \result[k] == xs[k]);
  pure seq<int> subseq(seq<int> xs, int i) = (i != |xs|-1) ? seq<int>{xs[i]} + subseq(xs, i+1) : seq<int>{}; 
@*/

/* ** Invariant 12 */
/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires 0 <= vertex && vertex < V;
requires Path(V, A, start, end, weight, src, vertex, P);
requires |P| <= i + 1;
requires oldcost[src] == 0 && cost[src] == 0 && cost[vertex] == inf();
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf());
requires (\forall int a; 0 <= a && a < A && oldcost[start[a]] != inf(); cost[end[a]] != inf());
ensures false;
pure bool lemma_inv12_helper1(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i, int vertex, seq<int> P) =
  vertex != src && 0 < |P| && Path(V, A, start, end, weight, src, start[P[|P| - 1]], subseq(P, 0));
@*/

/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires 0 <= vertex && vertex < V;
requires oldcost[src] == 0 && cost[src] == 0 && cost[vertex] == inf();
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf());
requires (\forall int a; 0 <= a && a < A && oldcost[start[a]] != inf(); cost[end[a]] != inf());
ensures (\forall seq<int> P; Path(V, A, start, end, weight, src, vertex, P) && |P| <= i + 1; false);
pure bool lemma_inv12_helper2(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i, int vertex) =
  (\forall seq<int> P; Path(V, A, start, end, weight, src, vertex, P) && |P| <= i + 1; lemma_inv12_helper1(V, A, start, end, weight, oldcost, cost, src, i, vertex, P));
@*/

/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires oldcost[src] == 0 && cost[src] == 0;
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf());
requires (\forall int a; 0 <= a && a < A && oldcost[start[a]] != inf(); cost[end[a]] != inf());
ensures (\forall int vertex; 0 <= vertex && vertex < V; (\forall seq<int> P; Path(V, A, start, end, weight, src, vertex, P) && |P| <= i + 1; vertex != inf()));
pure bool lemma_inv12_helper3(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i) =
  (\forall int vertex; 0 <= vertex && vertex < V && cost[vertex] == inf(); lemma_inv12_helper2(V, A, start, end, weight, oldcost, cost, src, i, vertex));
@*/

/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires oldcost[src] == 0 && cost[src] == 0;
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf());
requires (\forall int a; 0 <= a && a < A && oldcost[start[a]] != inf(); cost[end[a]] != inf());
ensures (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i + 1); cost[v] != inf());
pure bool lemma_inv12_preserved(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i) =
   lemma_inv12_helper3(V, A, start, end, weight, oldcost, cost, src, i);
@*/

/* ** Invariant 11 */
/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires 0 <= vertex && vertex < V;
requires Path(V, A, start, end, weight, src, vertex, P);
requires |P| <= i + 1;
requires oldcost[src] == 0 && cost[src] == 0 && cost[vertex] != inf();
requires cost(V, A, start, end, weight, P) < cost[vertex];
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); 
          (\forall seq<int> t; Path(V, A, start, end, weight, src, v, t) && 
           |t| <= i; oldcost[v] <= cost(V, A, start, end, weight, t))); // inv 11
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf()); // inv 12
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); ExPathEqual(V, A, start, end, weight, src, v, oldcost[v])); // inv 10
requires (\forall int a; 0 <= a && a < A && (oldcost[start[a]] != inf()) && 
           (oldcost[end[a]] == inf() || oldcost[start[a]] + weight[a] <= oldcost[end[a]]); cost[end[a]] <= oldcost[start[a]] + weight[a]);
ensures false;
pure bool lemma_inv11_helper1(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i, int vertex, seq<int> P) =
     |P| > 0 &&   
     (P == subseq(P, 0) + seq<int>{P[|P| - 1]}) &&
     lemma_cost_path(V, A, start, end, weight, src, start[P[|P| - 1]], vertex, subseq(P, 0), seq<int>{P[|P| - 1]});
@*/

/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires 0 <= vertex && vertex < V;
requires oldcost[src] == 0 && cost[src] == 0 && cost[vertex] != inf();
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); 
          (\forall seq<int> t; Path(V, A, start, end, weight, src, v, t) && 
           |t| <= i; oldcost[v] <= cost(V, A, start, end, weight, t))); // inv 11
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf()); // inv 12
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); ExPathEqual(V, A, start, end, weight, src, v, oldcost[v])); // inv 10
requires (\forall int a; 0 <= a && a < A && (oldcost[start[a]] != inf()) && 
           (oldcost[end[a]] == inf() || oldcost[start[a]] + weight[a] <= oldcost[end[a]]); cost[end[a]] <= oldcost[start[a]] + weight[a]);
ensures (\forall seq<int> P; Path(V, A, start, end, weight, src, vertex, P) && |P| <= i + 1 && cost(V, A, start, end, weight, P) < cost[vertex]; false);
pure bool lemma_inv11_helper2(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i, int vertex) =
  (\forall seq<int> P; Path(V, A, start, end, weight, src, vertex, P) && |P| <= i + 1 && cost(V, A, start, end, weight, P) < cost[vertex];
    lemma_inv11_helper1(V, A, start, end, weight, oldcost, cost, src, i, vertex, P));
@*/

/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires oldcost[src] == 0 && cost[src] == 0;
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); 
          (\forall seq<int> t; Path(V, A, start, end, weight, src, v, t) && 
           |t| <= i; oldcost[v] <= cost(V, A, start, end, weight, t))); // inv 11
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf()); // inv 12
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); ExPathEqual(V, A, start, end, weight, src, v, oldcost[v])); // inv 10
requires (\forall int a; 0 <= a && a < A && (oldcost[start[a]] != inf()) && 
           (oldcost[end[a]] == inf() || oldcost[start[a]] + weight[a] <= oldcost[end[a]]); cost[end[a]] <= oldcost[start[a]] + weight[a]);
ensures (\forall int vertex; 0 <= vertex && vertex < V && cost[vertex] != inf();
            (\forall seq<int> P; Path(V, A, start, end, weight, src, vertex, P) && |P| <= i + 1 && cost(V, A, start, end, weight, P) < cost[vertex]; vertex != inf()));
pure bool lemma_inv11_helper3(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i) =
  (\forall int vertex; 0 <= vertex && vertex < V && cost[vertex] != inf(); lemma_inv11_helper2(V, A, start, end, weight, oldcost, cost, src, i, vertex));
@*/

/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires oldcost[src] == 0 && cost[src] == 0;
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); 
          (\forall seq<int> t; Path(V, A, start, end, weight, src, v, t) && 
           |t| <= i; oldcost[v] <= cost(V, A, start, end, weight, t))); // inv 11
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf()); // inv 12
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); ExPathEqual(V, A, start, end, weight, src, v, oldcost[v])); // inv 10
requires (\forall int a; 0 <= a && a < A && (oldcost[start[a]] != inf()) && 
           (oldcost[end[a]] == inf() || oldcost[start[a]] + weight[a] <= oldcost[end[a]]); cost[end[a]] <= oldcost[start[a]] + weight[a]);
ensures (\forall int vertex; 0 <= vertex && vertex < V && cost[vertex] != inf();
            (\forall seq<int> P; Path(V, A, start, end, weight, src, vertex, P) && |P| <= i + 1; cost(V, A, start, end, weight, P) >= cost[vertex]));
pure bool lemma_inv11_preserved(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i) = 
  lemma_inv11_helper3(V, A, start, end, weight, oldcost, cost, src, i);
@*/

//////////////////////////////////////////////////////////////////////////////////Lemma1_simple_path
/*@
requires bound >= 0;
pure bool inbounds(seq<int> xs, int bound) = (\forall int i; i >= 0 && i < |xs|; xs[i] < bound);
@*/

/*@
pure bool nodupl(seq<int> xs) = 
(\forall int i; i >= 0 && i < |xs|; (\forall int j; j > i && j < |xs|; xs[i] != xs[j]));
@*/

/**
 * Gives the set containing all elements ranging from 0 up to (and including) `max`
 */
/*@
requires max >= 0;
ensures (\forall int v; v >= 0 && v <= max; (v \in \result));
ensures (\forall int v; (v \in \result); v >= 0 && v <= max);
ensures |\result| == max + 1;
pure set<int> rangeset(int max) = max > 0 ? set<int>{max} + rangeset(max-1) : set<int>{0};
@*/

/**
 * Converts `xs` to a set.
 */
/*@ 
ensures (\forall int v; (v \in \result); (v \in xs));
ensures (\forall int v; (v \in xs); (v \in \result));
pure set<int> seq2set(seq<int> xs) = |xs| > 0 ? set<int>{xs[0]} + seq2set(tail(xs)) : set<int>{};
@*/

/**
 * If all elements in `xs` are unique, then `xs` has the same size as `seq2set(xs)`.
 */
/*@
requires nodupl(xs);
ensures \result && |xs| == |seq2set(xs)|;
pure bool seq2set_nodupl(seq<int> xs) = |xs| > 0 ==> seq2set_nodupl(tail(xs));
@*/

/**
 * If all elements in `xs` are unique and within 0..`bound`,
 * then `xs` contains at most `bound` elements.
 */
/*@
requires 0 < bound;
requires inbounds(xs, bound);
requires nodupl(xs);
ensures \result;
pure bool seq_bounded_size(seq<int> xs, int bound) = 
  |seq2set(xs) - rangeset(bound - 1)| == |seq2set(xs)| - |seq2set(xs) * rangeset(bound - 1)| && seq2set_nodupl(xs);
@*/

/**
 * Gives the list of all vertices that are on `P`.
 * Since `P` is simple, all these vertices must be unique.
 */
/*@
requires Graph(V, A, start, end, weight);
requires SimplePath(V, A, start, end, weight, x, y, P);
ensures |\result| == |P| + 1;
ensures \result[0] == x;
ensures (\forall int i; 0 < i && i < |\result|; \result[i] == end[P[i - 1]]);
ensures inbounds(\result, V);
ensures nodupl(\result);
pure seq<int> collect(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) =
  seq<int>{x} + collect_helper(V, A, start, end, weight, x, y, P);
@*/

/**
 * Auxiliary helper function for `collect`.
 */
/*@
requires Graph(V, A, start, end, weight);
requires SimplePath(V, A, start, end, weight, x, y, P);
ensures |\result| == |P|;
ensures (\forall int i; 0 <= i && i < |\result|; \result[i] == end[P[i]]);
ensures inbounds(\result, V);
ensures nodupl(\result);
pure seq<int> collect_helper(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) =
  0 < |P| ? seq<int>{end[P[0]]} + collect_helper(V, A, start, end, weight, end[P[0]], y, tail(P)) : seq<int>{};
@*/

/*@
requires Graph(V, A, start, end, weight);
requires SimplePath(V, A, start, end, weight, x, y, P);
ensures \result && |P| < V;  
pure bool simple_path_lemma1_helper(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) =
  seq_bounded_size(collect(V, A, start, end, weight, x, y, P), V);
@*/

/*@
requires Graph(V, A, start, end, weight);
ensures (\forall seq<int> P; SimplePath(V, A, start, end, weight, x, y, P); |P| < V);
pure bool simple_path_lemma1(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y) =  
  (\forall seq<int> P; SimplePath(V, A, start, end, weight, x, y, P); simple_path_lemma1_helper(V, A, start, end, weight, x, y, P));
@*/

/*@
requires Graph(V, A, start, end, weight);
ensures \result && (\forall int v; v >= 0 && v < V; (\forall seq<int> P; SimplePath(V, A, start, end, weight, x, v, P); |P| < V));
pure bool simple_path_lemma1_preserved(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x) = 
  (\forall int v; v >= 0 && v < V; (\forall seq<int> P; SimplePath(V, A, start, end, weight, x, v, P); simple_path_lemma1(V, A, start, end, weight, x, v)) ); 
@*/

//////////////////////////////////////////////////////////////////////////////////Lemma2&3&4_simple_path
/**
  * Given any simple (x,y)-path `P` and simple (y,z)-path `Q`
  * for which it holds that `P[1..] ++ Q` is also a simple path (see fourth requires clause),
  * this function constructs and returns a simple (x,z)-path.
  */
/*@
requires Graph(V, A, start, end, weight);
requires SimplePath(V, A, start, end, weight, x, y, P);
requires SimplePath(V, A, start, end, weight, y, z, Q);
requires (\forall int i; i > 0 && i < |P|; 
          (\forall int j; j >= 0 && j < |Q|; start[P[i]] != end[Q[j]]));
requires (\forall int i; i >= 0 && i < |P|; end[P[i]] != x);
ensures SimplePath(V, A, start, end, weight, x, z, \result);
ensures |\result| <= |P| + |Q|;
ensures cost(V, A, start, end, weight, \result) <= cost(V, A, start, end, weight, P) + cost(V, A, start, end, weight, Q);
pure seq<int> trim(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, int z, seq<int> P, seq<int> Q) =
    0 < |Q| ? (end[Q[0]] == x ?
               trim(V, A, start, end, weight, x, x, z, seq<int>{}, tail(Q)) :
               assertSeq(trim(V, A, start, end, weight, x, end[Q[0]], z, P+seq<int>{Q[0]}, tail(Q)), 
               (lemma_cost_app(V, A, start, end, weight, P+seq<int>{Q[0]}, tail(Q)) &&
               lemma_app_append_right(P, Q) && 
               lemma_cost_app(V, A, start, end, weight, P, Q)))) 
            : P;
@*/  

/**
 * Given any simple (y,z)-path `P` and (x,y)-arc `a`,
 * this function constructs and returns a simple (x,z)-path.
 */
/*@
requires Graph(V, A, start, end, weight);
requires SimplePath(V, A, start, end, weight, y, z, P);
requires 0 <= a && a < A && start[a] == x && end[a] == y;
ensures SimplePath(V, A, start, end, weight, x, z, \result);
ensures |\result| <= |P| + 1; 
ensures cost(V, A, start, end, weight, \result) <= weight[a] + cost(V, A, start, end, weight, P);
pure  seq<int> extend(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, int z, int a, seq<int> P) =
   trim(V, A, start, end, weight, x, y, z, seq<int>{a}, P);
@*/

/**
 * Converts any given (x,y)-path `P` to a simple (x,y)-path.
 * The other Half part of lemma 2
 * And lemma 3 And lemma 4
 */
/*@  
requires Graph(V, A, start, end, weight);
requires Path(V, A, start, end, weight, x, y, P);
ensures SimplePath(V, A, start, end, weight, x, y, \result);
ensures |\result| <= |P|;
ensures cost(V, A, start, end, weight, \result) <= cost(V, A, start, end, weight, P);
pure seq<int> convert(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) =
  |P| > 0 ? extend(V, A, start, end, weight, x, end[P[0]], y, P[0], convert(V, A, start, end, weight, end[P[0]], y, tail(P)))
  : P;
@*/

/*@ 
requires Graph(V, A, start, end, weight);
requires SimplePath(V, A, start, end, weight, x, y, P);
ensures \result && Path(V, A, start, end, weight, x, y, P);    
pure bool simple_path_Part_1_lemma2_helper(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) = true;
@*/

/*@
requires Graph(V, A, start, end, weight);
ensures (\forall seq<int> P; SimplePath(V, A, start, end, weight, x, y, P); Path(V, A, start, end, weight, x, y, P));
pure bool simple_path_Part_1_lemma2(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y) =
(\forall seq<int> P; SimplePath(V, A, start, end, weight, x, y, P); simple_path_Part_1_lemma2_helper(V, A, start, end, weight, x, y, P));
@*/


/**
 * Every simple path is a path (trivial).
 */
/*@
requires Graph(V, A, start, end, weight);
ensures \result && (\forall int v; v >= 0 && v < V; 
         (\forall seq<int> P; SimplePath(V, A, start, end, weight, x, v, P); Path(V, A, start, end, weight, x, v, P)));
pure bool simple_path_Part_1_lemma2_preserved(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x) =
   (\forall int v; v >= 0 && v < V; simple_path_Part_1_lemma2(V, A, start, end, weight, x, v));
@*/

/**
 * For every path convert(path) is a path.
 * For every path convert(path) is a simple path.
 */
/*@
requires Graph(V, A, start, end, weight);
ensures (\forall int v; v >= 0 && v < V; 
         (\forall seq<int> P; Path(V, A, start, end, weight, x, v, P); Path(V, A, start, end, weight, x, v, convert(V, A, start, end, weight, x, v, P))));
ensures \result && (\forall int v; v >= 0 && v < V; 
         (\forall seq<int> P; Path(V, A, start, end, weight, x, v, P); SimplePath(V, A, start, end, weight, x, v, convert(V, A, start, end, weight, x, v, P))));
pure bool simple_path_Part_2_lemma2_preserved(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x) = true;
@*/

/**
 * For every path |convert(path)| <= |path|.
 */
/*@
requires Graph(V, A, start, end, weight);
ensures \result && (\forall int v; v >= 0 && v < V; 
         (\forall seq<int> P; Path(V, A, start, end, weight, x, v, P); 
           |convert(V, A, start, end, weight, x, v, P)| <= |P|));
pure bool simple_path_lemma3_preserved(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x) = true;
@*/

/**
 * For every path cost(convert(path)) <= cost(path).
 */
/*@
requires Graph(V, A, start, end, weight);
ensures \result && (\forall int v; v >= 0 && v < V; 
         (\forall seq<int> P; Path(V, A, start, end, weight, x, v, P); 
            cost(V, A, start, end, weight, convert(V, A, start, end, weight, x, v, P)) <= cost(V, A, start, end, weight, P)));
pure bool simple_path_cost_lemma4(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x) = true;
@*/

////////////////////////////////////////////////////////////////////////////////
//Array to Seq
////////////////////////////////////////////////////////////////////////////////
/*@ 
 requires \pointer(array, len, 1\2);
 requires i >= 0 && i <= len;
 ensures |\result| == len - i;
 ensures (\forall int j; j >= 0 && j < len - i; \result[j] == array[j+i]);
 pure seq<int> ArrayToSeq(pointer<int> array, int i, int len) = 
    i < len ? seq<int>{array[i]} + ArrayToSeq(array, i+1, len) : seq<int>{};  
@*/
////////////////////////////////////////////////////////////////////////////////
//compare
////////////////////////////////////////////////////////////////////////////////  
  /*@
  requires start_cost == inf() || start_cost >= 0;
  requires end_cost == inf() || end_cost >= 0;
  requires weight > 0;
  ensures (start_cost != inf()) && (end_cost == inf() || start_cost + weight <= end_cost) ==> \result;
  ensures \result ==> (start_cost != inf()) && (end_cost == inf() || start_cost + weight <= end_cost);
  ensures (start_cost == inf()) || (end_cost != inf() && start_cost + weight > end_cost) ==> !\result;
  ensures !\result ==> (start_cost == inf()) || (end_cost != inf() && start_cost + weight > end_cost);
  ensures (\result && end_cost != inf()) ==> start_cost + weight <= end_cost;
  pure bool compare(int start_cost, int weight, int end_cost) = 
    (start_cost != inf() && (end_cost == inf() || start_cost+weight <= end_cost)) ? true : false;
  @*/
////////////////////////////////////////////////////////////////////////////////
//Kernel
////////////////////////////////////////////////////////////////////////////////

/*@
  context_everywhere V == 1024 && A == 10 * V;
  context_everywhere opencl_gsize == V;
  context_everywhere 10 == opencl_gcount;
  context_everywhere source >= 0 && source < V;
  context_everywhere counter >= 0 && counter < V-1;

  context \pointer_index(g_contrib, \gtid, 1\2);
  requires g_contrib[\gtid] == 0;
  ensures g_contrib[\gtid] == 1;

  kernel_invariant \pointer(g_start, A, 1\2);
  kernel_invariant \pointer(g_end, A, 1\2);
  kernel_invariant (\forall int i; 0 <= i && i < A; g_start[i] >= 0 && g_start[i] < V && g_end[i] >= 0 && g_end[i] < V);
  kernel_invariant (\forall int i; 0 <= i && i < A; g_start[i] != g_end[i]);
  kernel_invariant (\forall int i; 0 <= i && i < A; (\forall int j; 0 <= j && j < A && i != j; g_start[i] == g_start[j] ==> g_end[i] != g_end[j]));
  
  kernel_invariant \pointer(g_weight, A, 1\2);
  kernel_invariant (\forall int i; 0 <= i && i < A; g_weight[i] > 0);
  
  kernel_invariant \pointer(g_cost, V, 1);
  kernel_invariant (\forall int i; 0 <= i && i < V && i != source; g_cost[i] == inf() || g_cost[i] > 0); 
  kernel_invariant g_cost[source] == 0;

  kernel_invariant \pointer(g_old_cost, V, 1);
 
  kernel_invariant \pointer(g_contrib, A, 1\2);

  
@*/
__global__ void CUDAKernel(int* g_start, int* g_end, int* g_weight, int* g_cost, int* g_old_cost, int* g_contrib, int V, int A, int counter, int source)
{
  int tid = blockIdx.x * V + threadIdx.x;
  //@ assert tid == \gtid;
  atomicRelax(g_cost+g_end[tid], g_weight[tid], g_cost[g_start[tid]]) /*@ then { g_contrib[tid] = 1; } */;

}


////////////////////////////////////////////////////////////////////////////////
//@ ensures \pointer(\result, N, write);
int *vercorsMallocInt(int N);
void vercorsFreeInt(int *ar);
//@ ensures \pointer(\result, N, write);
int *vercorsCudaMallocInt(int N);
void vercorsCudaFreeInt(int *addr);
void vercorsCudaMemcpyInt(int *tgt, int *src, int N, int direction); 

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) 
{
	
	int V = 1024; // no. of vertices
	int A = 10 * V; // no. of arcs
    	
	// allocate host memory
  int* host_start = vercorsMallocInt(A);
  int* host_end = vercorsMallocInt(A);
  int* host_weight = vercorsMallocInt(A);
  int* host_cost = vercorsMallocInt(V);
  int* host_old_cost = vercorsMallocInt(V);
  int* host_contrib = vercorsMallocInt(A);
  
  int source = V/2;
  
  // these assumptions come from the input graph file which we change it into assertions 
  //@ assume (\forall int i; 0 <= i && i < A; host_start[i] >= 0 && host_start[i] < V);
  //@ assume (\forall int i; 0 <= i && i < A; host_end[i] >= 0 && host_end[i] < V);
  //@ assume (\forall int i; 0 <= i && i < A; host_start[i] != host_end[i]); // No self-loop
  
  //@ assume (\forall int i; 0 <= i && i < A; host_weight[i] > 0);
  
  //@ assume (\forall int i; 0 <= i && i < A; (\forall int j; 0 <= j && j < A && i != j; host_start[i] == host_start[j] ==> host_end[i] != host_end[j])); 
  
  //@ loop_invariant i >= 0 && i <= V;
  //@ loop_invariant (\forall* int k; 0 <= k && k < V; Perm(host_cost[k], write));
  //@ loop_invariant (\forall int k; 0 <= k && k < i; host_cost[k] == -1);   
  for(unsigned int i = 0; i < V; i++) 
  {
    host_cost[i] = -1;
  }
  host_cost[source] = 0;
  //host_old_cost = host_cost;
  
  //@ loop_invariant g >= 0 && g <= V;
  //@ loop_invariant (\forall* int k; 0 <= k && k < V; Perm(host_cost[k], write));
  //@ loop_invariant (\forall* int k; 0 <= k && k < V; Perm(host_old_cost[k], write));
  //@ loop_invariant (\forall int k; 0 <= k && k < g; host_cost[k] == host_old_cost[k]);   
  for(unsigned int g = 0; g < V; g++) 
  {
    host_cost[g] = host_old_cost[g];
  }
  
  //@ assert (\forall int i; 0 <= i && i < V; host_old_cost[i] == host_cost[i]);
  
  //@ assert (\forall int i; 0 <= i && i < V && i != source; host_cost[i] == inf());
  
  //@ assert host_cost[source] == 0;
  
  //@ loop_invariant r >= 0 && r <= A;
  //@ loop_invariant \pointer(host_contrib, A, 1);
  //@ loop_invariant (\forall int k; 0 <= k && k < r; host_contrib[k] == 0);   
  for(unsigned int r = 0; r < A; r++) 
  {
    host_contrib[r] = 0;
  }

  //Copy the arrays to device memory
  int* device_start;
  device_start = vercorsCudaMallocInt(A);
  vercorsCudaMemcpyInt( device_start, host_start, A, hipMemcpyHostToDevice) ;
  //@ assume (\forall int i; i >= 0 && i < A; host_start[i] == device_start[i]);

  //Copy the arrays to device memory
  int* device_end;
  device_end = vercorsCudaMallocInt(A);
  vercorsCudaMemcpyInt( device_end, host_end, A, hipMemcpyHostToDevice) ;
  //@ assume (\forall int i; i >= 0 && i < A; host_end[i] == device_end[i]);
  
  //Copy the arrays to device memory
  int* device_weight;
  device_weight = vercorsCudaMallocInt(A);
  vercorsCudaMemcpyInt( device_weight, host_weight, A, hipMemcpyHostToDevice) ;
  //@ assume (\forall int i; i >= 0 && i < A; host_weight[i] == device_weight[i]);


  //Copy the arrays to device memory
  int* device_cost;
  device_cost = vercorsCudaMallocInt(V);
  vercorsCudaMemcpyInt( device_cost, host_cost, V, hipMemcpyHostToDevice) ;
  //@ assume (\forall int i; i >= 0 && i < V; host_cost[i] == device_cost[i]);
  
  //Copy the arrays to device memory
  int* device_old_cost;
  device_old_cost = vercorsCudaMallocInt(V);
  
  //@ loop_invariant m >= 0 && m <= V;
  //@ loop_invariant \pointer(device_old_cost, V, 1);
  //@ loop_invariant \pointer(device_cost, V, 1);
  //@ loop_invariant (\forall int k; 0 <= k && k < m; device_old_cost[k] == device_cost[k]);   
  for(unsigned int m = 0; m < V; m++) 
  {
    device_old_cost[m] = device_cost[m];
  }
  
  vercorsCudaMemcpyInt( device_old_cost, host_old_cost, V, hipMemcpyHostToDevice) ;
  //@ assume (\forall int i; i >= 0 && i < V; host_old_cost[i] == device_old_cost[i]);
  
  
  //Copy the arrays to device memory
  int* device_contrib;
  device_contrib = vercorsCudaMallocInt(A);
  
  //@ loop_invariant q >= 0 && q <= A;
  //@ loop_invariant \pointer(device_contrib, A, 1);
  //@ loop_invariant (\forall int k; 0 <= k && k < q; device_contrib[k] == 0);   
  for(unsigned int q = 0; q < A; q++) 
  {
    device_contrib[q] = 0;
  }
  
  vercorsCudaMemcpyInt( device_contrib, host_contrib, A, hipMemcpyHostToDevice) ;
  //@ assert (\forall int i; i >= 0 && i < A; device_contrib[i] == 0);
    
  //setup execution parameters
	int num_of_blocks = 10;
	int num_of_threads_per_block = V;
  
  //dim3  grid( num_of_blocks, 1, 1); //grid has three parameters to indicate the dimensions. Here we have one dimensional grid (of blocks). It can be one, two or three dimensions.
  //dim3  threads( num_of_threads_per_block, 1, 1); //threads indicates the dimensions of one block. Here each block has one dimension (of threads). It can be one, two or three dimensions.
  
  int counter = 0;


  //@ loop_invariant V == 1024 && A == 10 * V;
  //@ loop_invariant source >= 0 && source < V;
  //@ loop_invariant counter >= 0 && counter <= V - 1;
  //@ loop_invariant \pointer(device_start, A, 1\2); 
  //@ loop_invariant (\forall int i; 0 <= i && i < A; device_start[i] >= 0 && device_start[i] < V);
  //@ loop_invariant \pointer(device_end, A, 1\2);
  //@ loop_invariant (\forall int i; 0 <= i && i < A; device_end[i] >= 0 && device_end[i] < V);
  //@ loop_invariant (\forall int i; 0 <= i && i < A; device_start[i] != device_end[i]); 
  //@ loop_invariant (\forall int i; 0 <= i && i < A; (\forall int j; 0 <= j && j < A && i != j; device_start[i] == device_start[j] ==> device_end[i] != device_end[j]));		
  //@ loop_invariant \pointer(device_weight, A, 1\2);
  //@ loop_invariant (\forall int i; 0 <= i && i < A; device_weight[i] > 0);	
  //@ loop_invariant \pointer(device_old_cost, V, 1);						
  //@ loop_invariant \pointer(device_cost, V, 1);
  //@ loop_invariant (\forall int i; 0 <= i && i < V && i != source; device_cost[i] == inf() || device_cost[i] > 0);
  //@ loop_invariant device_cost[source] == 0; 
  //@ loop_invariant (\forall int i; i >= 0 && i < V; device_old_cost[i] == device_cost[i]);
  //@ loop_invariant \pointer(device_contrib, A, 1);	
  //@ loop_invariant (\forall int i; i >= 0 && i < A; device_contrib[i] == 0);							
  while(counter < V-1)
  {  
    //@ assert \pointer(device_old_cost, V, 1);
    
	  //Kernel launch
    CUDAKernel<<< /*grid*/num_of_blocks, /*threads*/num_of_threads_per_block/*, 0*/ >>>(device_start, device_end, device_weight, device_cost, device_old_cost, device_contrib, V, A, counter, source);

    //@ loop_invariant n >= 0 && n <= V;
    //@ loop_invariant \pointer(device_old_cost, V, 1);
    //@ loop_invariant \pointer(device_cost, V, 1);
    //@ loop_invariant (\forall int k; 0 <= k && k < n; device_old_cost[k] == device_cost[k]);   
    for(unsigned int n = 0; n < V; n++) 
    {
      device_old_cost[n] = device_cost[n];
    }
    //@ assert (\forall int i; i >= 0 && i < V; device_old_cost[i] == device_cost[i]);
    
    //@ loop_invariant h >= 0 && h <= A;
    //@ loop_invariant \pointer(device_contrib, A, 1);
    //@ loop_invariant (\forall int k; 0 <= k && k < h; device_contrib[k] == 0);   
    for(unsigned int h = 0; h < A; h++) 
    {
      device_contrib[h] = 0;
    }

    counter = counter + 1;
  }
  
  
  // copy result from device to host
  vercorsCudaMemcpyInt(host_cost, device_cost, V, hipMemcpyDeviceToHost);	
	 
  
  // cleanup memory
  vercorsFreeInt(host_start);
  vercorsFreeInt(host_end);
  vercorsFreeInt(host_weight);
  vercorsFreeInt(host_cost);
  vercorsCudaFreeInt(device_start);
  vercorsCudaFreeInt(device_end);
  vercorsCudaFreeInt(device_weight);
  vercorsCudaFreeInt(device_cost);

	return 0;
       
}

